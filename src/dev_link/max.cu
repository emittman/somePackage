#include "../iterator.h"
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <hipblas.h>
#include <iostream>

typedef thrust::tuple<strideAccessor, intIter> my_tuple;

struct whichMax : thrust::unary_function<double, int>{
  int dim;
  __host__ __device__ whichMax(int dim): dim(dim){}

  __host__ __device__ int operator()(double &vec){

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int incx=1, n = dim, result =0;
    double *vec_ptr = thrust::raw_pointer_cast(&vec);
    //find the first index of a maximal element
    hipblasIdamax(handle, n, vec_ptr, incx, &result);
    hipblasDestroy(handle);
    return result;
  }
};

void cublas_max(fvec_d &x, ivec_d &result, int n, int d){
  stride f(d);
  strideIter siter = thrust::transform_iterator<stride, countIter>(thrust::make_counting_iterator<int>(0), f);
  strideAccessor stridex = thrust::permutation_iterator<realIter, strideIter>(x.begin(), siter);

  whichMax g(d);

  //find the index of maximum for each of n subvectors
  thrust::copy(result.begin(), result.end(), std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  thrust::transform(stridex, stridex + n, result.begin(),  g);
  thrust::copy(result.begin(), result.end(), std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
}
