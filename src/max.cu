#include "iterator.h"
#include <thrust/functional.h>
#include <hipblas.h>

typedef thrust::tuple<strideAccessor, intIter> my_tuple;
typedef thrust::zip_iterator<my_tuple> my_zip;
typedef thrust::tuple<double&, int&> el_tuple;

struct whichMax : thrust::unary_function<el_tuple &, void>{
  int dim;
  __host__ __device__ whichMax(int dim): dim(dim){}

  __device__ void operator()(el_tuple &Tup){

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int incx=1, n = dim;
    double *x = thrust::raw_pointer_cast(&(thrust::get<0>(Tup)));
    double *result = thrust::raw_pointer_cast(&(thrust::get<1>(Tup)));
    //find the first index of a maximal element
    hipblasIdamax(handle, x, incx, result)
    hipblasDestroy(handle);
  }
};

void cublas_max(fvec_d x, ivec_d result, int n, int d){
  stride f(d);
  strideIter siter = thrust::transform_iterator<stride, countIter>(thrust::make_counting_iterator<int>(0), f);
  strideAccessor stridex = thrust::permutation_iterator<realIter, strideIter>(x.begin(), siter);
  my_tuple tup = thrust::tuple<strideAccessor, intIter>(stridex, result.begin());
  my_zip zip = thrust::zip_iterator<my_tup>(tup);
  whichMax g(d);
  //find the index of maximum for each of n subvectors
  thrust::for_each(zip, zip + n, g);
}
