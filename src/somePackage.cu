#include "hip/hip_runtime.h"
#include "header/iterator.h"
#include "header/max.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <R.h>
#include <Rinternals.h>
#include <Rmath.h>
#include <iostream>

extern "C" SEXP Rcublas_max(SEXP x, SEXP n, SEXP dim){

  double *xptr = REAL(x);
  int N = INTEGER(n)[0], D = INTEGER(dim)[0];

  fvec_d dx(xptr, xptr+N*D);
  ivec_d dresult(N);

  cublas_max(dx, dresult, N, D);

  ivec_h hresult(N);
  thrust::copy(dresult.begin(), dresult.end(), hresult.begin());

  SEXP indices = PROTECT(allocVector(INTSXP, N));

  for(int i=0; i<N; ++i)
    INTEGER(indices)[i] = hresult[i];

  UNPROTECT(1);
  return indices;
}


