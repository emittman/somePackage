
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmult(const double *A, const double *B, double *C, const int m, const int k, const int n) {
  int lda=m,ldb=k,ldc=m;
  const double alf = 1;
  const double bet = 0;
  const double *alpha = &alf;
  const double *beta = &bet;
  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  // Do the actual multiplication
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  // Destroy the handle
  hipblasDestroy(handle);
}
